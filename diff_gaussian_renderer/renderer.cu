#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <vector>
#include <iostream>
#include <cstdio>
#include <fstream>
#include <optix.h>
#include <optix_function_table_definition.h>
#include <optix_stubs.h>
#include <optix_types.h>
#include <optix_host.h>

#include "sutil/Camera.h"

struct Params
{
    uchar4*  image;
    unsigned int  image_width;
    unsigned int  image_height;
    float3   cam_eye;
    float3   cam_u, cam_v, cam_w;
    OptixTraversableHandle handle;
};
// These structs represent the data blocks of our SBT records
struct RayGenData   { };// No data needed };
struct HitGroupData { };// No data needed };
struct MissData     { float3 bg_color;  };

// SBT record with an appropriately aligned and sized data block
template <typename T>
struct SbtRecord
{
    __align__( OPTIX_SBT_RECORD_ALIGNMENT )
    char header[OPTIX_SBT_RECORD_HEADER_SIZE];
    T data;
};

typedef SbtRecord<RayGenData>   RayGenSbtRecord;
typedef SbtRecord<MissData>     MissSbtRecord;
typedef SbtRecord<HitGroupData> HitGroupSbtRecord;

#define OPTIX_CHECK(error)                                                     \
  {                                                                            \
    if (error != OPTIX_SUCCESS)                                                \
      std::cerr << __FILE__ << ":" << __LINE__ << " Optix Error: '"            \
                << optixGetErrorString(error) << "'\n";                        \
  }

#define CUDA_CHECK(error)                                                      \
{                                                                            \
    if (error != hipSuccess)                                                  \
      std::cerr << __FILE__ << ":" << __LINE__ << " CUDA Error: '"             \
                << hipGetErrorString(error) << "'\n";                         \
}

void optixLogCallback(unsigned int level, const char *tag, const char *message,
                      void *cbdata) {
  std::cout << "Optix Log[" << level << "][" << tag << "]: '" << message
            << "'\n";
}

OptixDeviceContext createOptixContext() {
  hipFree(0); // creates a CUDA context if there isn't already one
  optixInit(); // loads the optix library and populates the function table

  OptixDeviceContextOptions options = {};
  options.logCallbackFunction = &optixLogCallback;
  options.logCallbackLevel = 4;

  OptixDeviceContext optix_context = nullptr;
  optixDeviceContextCreate(0, // use current CUDA context
                           &options, &optix_context);

  return optix_context;
}

std::string loadPtx(std::string filename) {
  std::ifstream ptx_in(filename);
  return std::string((std::istreambuf_iterator<char>(ptx_in)),
                     std::istreambuf_iterator<char>());
}

class OptixState {
public:

    // Constructors
    OptixState() {
        std::cout << "Creating optix context" << std::endl;
        init_context();

        std::cout << "Building gas" << std::endl;
        build_gas();

        std::cout << "Building ias" << std::endl;
        build_ias();

        std::cout << "Building module" << std::endl;
        build_module();

        std::cout << "Building pipeline and sbt" << std::endl;
        // TODO: Change image width and height to be passed in
        build_pipeline_and_sbt(2560, 1440);

        
    }

    // Destructors
    // ~OptixState() {
    //     hipFree((void*)d_output);
    //     hipFree((void*)d2_output);
    // }

    // Functions
    // TODO: Refactor to build gaussians around multiple gaussians
    void build_ias() {
        OptixInstance instance = {};
        float transform[12] = {0.7071,-0.7071,0.0,0,0.7071,0.7071,0.0 ,0,0.0,0.0,1.0,0};
        memcpy( instance.transform, transform, sizeof( float )*12 );
        instance.instanceId = 0;
        instance.visibilityMask = 255;
        instance.sbtOffset = 0;
        instance.flags = OPTIX_INSTANCE_FLAG_NONE;
        instance.traversableHandle = gas_handle;

        hipDeviceptr_t d_instance;
        hipMalloc( (void**) &d_instance, sizeof( OptixInstance ) );
        hipMemcpy( (void*)d_instance, &instance, 
            sizeof( OptixInstance ), hipMemcpyHostToDevice );

        // Reset build input and bufferSizes
        OptixBuildInput buildInput = {};
        buildInput.type = OPTIX_BUILD_INPUT_TYPE_INSTANCES;
        OptixBuildInputInstanceArray& instanceArray = buildInput.instanceArray;
        instanceArray.instances = d_instance;
        instanceArray.numInstances = 1;
        
        hipStream_t streamDefault  = 0;
        OptixAccelBuildOptions accelOptions = {};
        accelOptions.buildFlags = OPTIX_BUILD_FLAG_NONE;
        accelOptions.operation = OPTIX_BUILD_OPERATION_BUILD;

        std::cout << "Computing memory usage" << std::endl;
        OptixAccelBufferSizes bufferSizes = {};
        optixAccelComputeMemoryUsage(context, &accelOptions,
            &buildInput, 1, &bufferSizes);

        std::cout << "Allocating memory" << std::endl;
        
        hipDeviceptr_t d2_temp;

        std::printf("output size: %llu\n", bufferSizes.outputSizeInBytes);
        std::printf("temp size: %llu\n", bufferSizes.tempSizeInBytes);
        hipMalloc((void**)&d2_output, bufferSizes.outputSizeInBytes);
        hipMalloc((void**)&d2_temp, bufferSizes.tempSizeInBytes);

        OptixTraversableHandle inHandle = 1;

        std::cout << "Building instance acceleration structure" << std::endl;
        // Build GAS Timer
        std::chrono::high_resolution_clock::time_point build_ias_start = std::chrono::high_resolution_clock::now();
        OptixResult results = optixAccelBuild(context, streamDefault,
            &accelOptions, &buildInput, 1, d2_temp,
            bufferSizes.tempSizeInBytes, d2_output,
            bufferSizes.outputSizeInBytes, &inHandle, nullptr, 0);
        std::chrono::high_resolution_clock::time_point build_ias_end = std::chrono::high_resolution_clock::now();
        std::cout << "Build ias time taken: " << std::chrono::duration_cast<std::chrono::milliseconds>(build_ias_end - build_ias_start).count() << " ms" << std::endl;
        instanceHandle = inHandle;
        hipFree((void*)d2_temp);
    }

    // Variables
    OptixDeviceContext context;

    OptixTraversableHandle gas_handle;
    hipDeviceptr_t d_output;
    OptixTraversableHandle instanceHandle;
    hipDeviceptr_t d2_output;

    OptixModule module;
    OptixPipelineCompileOptions pipeline_compile_options;
    OptixShaderBindingTable sbt;

    OptixPipeline pipeline;

    hipDeviceptr_t d_image;

private:
    void init_context() {
        
        context = createOptixContext();
    }

    void build_gas() {
        std::vector<float> vertex_buffer = {
            -0.5f, -0.5f, 0.0f,
            -0.5f,  0.5f, 0.0f,
            0.5f, -0.5f, 0.0f,
            0.5f,  0.5f, 0.0f
        };

        std::vector<uint32_t> index_buffer = {
            0, 1, 2,
            1, 2, 3
        };

        hipDeviceptr_t vertex_device;
        hipMalloc((void**)&vertex_device, sizeof(float) * vertex_buffer.size());
        hipMemcpy((void*)vertex_device, vertex_buffer.data(), sizeof(float) * vertex_buffer.size(), hipMemcpyHostToDevice);

        hipDeviceptr_t index_device;
        hipMalloc((void**)&index_device, sizeof(uint32_t) * index_buffer.size());
        hipMemcpy((void*)index_device, index_buffer.data(), sizeof(uint32_t) * index_buffer.size(), hipMemcpyHostToDevice);

        const uint32_t triangle_input_flags[1] = { OPTIX_GEOMETRY_FLAG_NONE };

        std::cout << "Creating build input" << std::endl;
        OptixBuildInput buildInput = {};
        buildInput.type = OPTIX_BUILD_INPUT_TYPE_TRIANGLES;
        // Create optix build input for triangles
        OptixBuildInputTriangleArray& triangleArray = buildInput.triangleArray;
        triangleArray.vertexBuffers = &vertex_device;
        triangleArray.numVertices = vertex_buffer.size();
        triangleArray.vertexFormat = OPTIX_VERTEX_FORMAT_FLOAT3;
        triangleArray.vertexStrideInBytes = sizeof(float) * 3;
        triangleArray.indexBuffer = index_device;
        triangleArray.numIndexTriplets = index_buffer.size() / 3;
        triangleArray.indexFormat = OPTIX_INDICES_FORMAT_UNSIGNED_INT3; 
        triangleArray.indexStrideInBytes = sizeof(int) * 3;
        triangleArray.preTransform = 0;
        triangleArray.numSbtRecords = 1;
        triangleArray.flags = triangle_input_flags;

        hipStream_t streamDefault  = 0;
        OptixAccelBuildOptions accelOptions = {};
        accelOptions.buildFlags = OPTIX_BUILD_FLAG_NONE;
        accelOptions.operation = OPTIX_BUILD_OPERATION_BUILD;

        std::cout << "Computing memory usage" << std::endl;
        OptixAccelBufferSizes bufferSizes = {};
        optixAccelComputeMemoryUsage(context, &accelOptions,
            &buildInput, 1, &bufferSizes);

        std::cout << "Allocating memory" << std::endl;
        
        hipDeviceptr_t d_temp;

        std::printf("output size: %llu\n", bufferSizes.outputSizeInBytes);
        std::printf("temp size: %llu\n", bufferSizes.tempSizeInBytes);
        hipMalloc((void**)&d_output, bufferSizes.outputSizeInBytes);
        hipMalloc((void**)&d_temp, bufferSizes.tempSizeInBytes);

        OptixTraversableHandle outputHandle = 1;
        std::cout << "Building gas acceleration structure" << std::endl;
        // Build GAS Timer
        std::chrono::high_resolution_clock::time_point build_gas_start = std::chrono::high_resolution_clock::now();
        OptixResult results = optixAccelBuild(context, streamDefault,
            &accelOptions, &buildInput, 1, d_temp,
            bufferSizes.tempSizeInBytes, d_output,
            bufferSizes.outputSizeInBytes, &outputHandle, nullptr, 0);
        std::chrono::high_resolution_clock::time_point build_gas_end = std::chrono::high_resolution_clock::now();
        std::cout << "Build gas time taken: " << std::chrono::duration_cast<std::chrono::milliseconds>(build_gas_end - build_gas_start).count() << " ms" << std::endl;

        if (results == OPTIX_SUCCESS) {
            std::cout << "Successfully built acceleration structure" << std::endl;
        } else {
            std::cout << "Failed to build acceleration structure" << std::endl;
        }

        gas_handle = outputHandle;
        hipFree((void*)d_temp);
    }

    void build_module() {
        
        pipeline_compile_options.usesMotionBlur = false;

        // This option is important to ensure we compile code which is optimal
        // for our scene hierarchy. We use a single GAS – no instancing or
        // multi-level hierarchies
        //pipeline_compile_options.traversableGraphFlags = OPTIX_TRAVERSABLE_GRAPH_FLAG_ALLOW_SINGLE_GAS;

        // Our device code uses 3 payload registers (r,g,b output value)
        pipeline_compile_options.numPayloadValues = 3;

        // This is the name of the param struct variable in our device code
        pipeline_compile_options.pipelineLaunchParamsVariableName = "params";
        
        std::string ptx = loadPtx("/home/teja/research/optix_splats/_skbuild/linux-x86_64-3.11/cmake-build/ptx/kernels.ptx");
        module = nullptr;
        OptixModuleCompileOptions module_compile_options = {};
        module_compile_options.maxRegisterCount =
            OPTIX_COMPILE_DEFAULT_MAX_REGISTER_COUNT;
        module_compile_options.optLevel = OPTIX_COMPILE_OPTIMIZATION_DEFAULT;
        module_compile_options.debugLevel = OPTIX_COMPILE_DEBUG_LEVEL_MODERATE;

        pipeline_compile_options.usesMotionBlur = false;
        pipeline_compile_options.traversableGraphFlags =
            OPTIX_TRAVERSABLE_GRAPH_FLAG_ALLOW_SINGLE_LEVEL_INSTANCING;
        pipeline_compile_options.numPayloadValues = 3;
        pipeline_compile_options.numAttributeValues = 2; // 2 is the minimum
        pipeline_compile_options.exceptionFlags = OPTIX_EXCEPTION_FLAG_NONE;
        pipeline_compile_options.pipelineLaunchParamsVariableName = "params";

        optixModuleCreate(context, &module_compile_options,
                                            &pipeline_compile_options, ptx.c_str(),
                                            ptx.size(), nullptr, nullptr, &module);
    }

    void build_pipeline_and_sbt(int image_width, int image_height) {
        OptixProgramGroup raygen_prog_group = nullptr;
        OptixProgramGroup miss_prog_group = nullptr;
        OptixProgramGroup hitgroup_prog_group = nullptr;

        OptixProgramGroupOptions program_group_options = {}; 
        OptixProgramGroupDesc raygen_prog_group_desc = {};
        raygen_prog_group_desc.kind = OPTIX_PROGRAM_GROUP_KIND_RAYGEN;
        raygen_prog_group_desc.raygen.module = module;
        raygen_prog_group_desc.raygen.entryFunctionName = "__raygen__rg";
        optixProgramGroupCreate(
            context,
            &raygen_prog_group_desc,
            1, // num program groups
            &program_group_options,
            nullptr,
            nullptr,
            &raygen_prog_group );
        
        OptixProgramGroupDesc miss_prog_group_desc = {};
        miss_prog_group_desc.kind = OPTIX_PROGRAM_GROUP_KIND_MISS;
        miss_prog_group_desc.miss.module = module;
        miss_prog_group_desc.miss.entryFunctionName = "__miss__ms";
        optixProgramGroupCreate(
            context,
            &miss_prog_group_desc,
            1, // num program groups
            &program_group_options,
            nullptr,
            nullptr,
            &miss_prog_group );
        
        OptixProgramGroupDesc hitgroup_prog_group_desc = {};
        hitgroup_prog_group_desc.kind = OPTIX_PROGRAM_GROUP_KIND_HITGROUP;
        hitgroup_prog_group_desc.hitgroup.moduleCH = module;
        hitgroup_prog_group_desc.hitgroup.entryFunctionNameCH = "__closesthit__ch";
        // We could also specify an any-hit and/or intersection program here
        optixProgramGroupCreate(
            context,
            &hitgroup_prog_group_desc,
            1, // num program groups
            &program_group_options,
            nullptr,
            nullptr,
            &hitgroup_prog_group );
        OptixProgramGroup program_groups[] = 
        { 
            raygen_prog_group, 
            miss_prog_group, 
            hitgroup_prog_group
        };
        
        OptixPipelineLinkOptions pipeline_link_options = {};
        pipeline_link_options.maxTraceDepth = 1;

        pipeline = nullptr;
        optixPipelineCreate(
            context,
            &pipeline_compile_options,
            &pipeline_link_options,
            program_groups,
            sizeof( program_groups ) / sizeof( program_groups[0] ),
            nullptr,
            nullptr,
            &pipeline );
        
        
        
        
        hipDeviceptr_t miss_record;
        size_t miss_record_size = sizeof( MissSbtRecord );
        hipMalloc( reinterpret_cast<void**>( &miss_record ), miss_record_size );

        hipDeviceptr_t raygen_record;
        size_t raygen_record_size = sizeof( RayGenSbtRecord );
        hipMalloc( reinterpret_cast<void**>( &raygen_record ), raygen_record_size );

        hipDeviceptr_t hitgroup_record;
        size_t hitgroup_record_size = sizeof( HitGroupSbtRecord );
        hipMalloc( reinterpret_cast<void**>( &hitgroup_record ), hitgroup_record_size );

        // Populate host-side copy of the record with header and data
        MissSbtRecord ms_sbt;
        ms_sbt.data.bg_color = { 0.3f, 0.1f, 0.2f };
        optixSbtRecordPackHeader( miss_prog_group, &ms_sbt );

        RayGenSbtRecord rg_sbt;
        optixSbtRecordPackHeader( raygen_prog_group, &rg_sbt );

        HitGroupSbtRecord hg_sbt;
        optixSbtRecordPackHeader( hitgroup_prog_group, &hg_sbt );

        // Now copy our host record to the device
        hipMemcpy(
            reinterpret_cast<void*>( miss_record ),
            &ms_sbt,
            miss_record_size,
            hipMemcpyHostToDevice );
        
        hipMemcpy(
            reinterpret_cast<void*>( raygen_record ),
            &rg_sbt,
            raygen_record_size,
            hipMemcpyHostToDevice );
        
        hipMemcpy(
            reinterpret_cast<void*>( hitgroup_record ),
            &hg_sbt,
            hitgroup_record_size,
            hipMemcpyHostToDevice );
        
        // The shader binding table struct we will populate
        sbt = {};

        // Finally we specify how many records and how they are packed in memory
        sbt.raygenRecord  = raygen_record;
        sbt.missRecordBase  = miss_record;
        sbt.missRecordStrideInBytes = sizeof( MissSbtRecord ); 
        sbt.missRecordCount  = 1;
        sbt.hitgroupRecordBase  = hitgroup_record;
        sbt.hitgroupRecordStrideInBytes = sizeof( HitGroupSbtRecord );
        sbt.hitgroupRecordCount  = 1;

        
        hipMalloc( reinterpret_cast<void**>( &d_image ),
            image_width * image_height * sizeof( uchar4 ) );
    }

    
    

};


torch::Tensor render_gaussians(int image_height, int image_width,
                               float camera_x, float camera_y, float camera_z,
                               float lookat_x, float lookat_y, float lookat_z,
                               float up_x, float up_y, float up_z,
                            //    float means_x, float means_y, float means_z,
                            //    float scales_x, float scales_y, float scales_z,
                            //    float rotations_a, float rotations_b, float rotations_c, float rotations_d, 
                               OptixState state) {

    
    

    std::cout << "Making image tensor height " << image_height << " width " << image_width << std::endl;
    // create torch tensor with size of image_height x image_width x 3 
    
    OptixDeviceContext context = state.context;

    OptixTraversableHandle instanceHandle = state.instanceHandle;
    OptixModule module = state.module;
      
    
    sutil::Camera cam;
    cam.setEye( {camera_x, camera_y, camera_z} );
    cam.setLookat( {lookat_x, lookat_y, lookat_z} );
    cam.setUp( {up_x, up_y, up_z} );
    cam.setFovY( 45.0f );
    cam.setAspectRatio( (float)image_width / (float)image_height );

    
    Params params;
    params.image_width = image_width;
    params.image_height = image_height;
    params.cam_eye      = cam.eye();
    params.handle = state.instanceHandle;
    cam.UVWFrame( params.cam_u, params.cam_v, params.cam_w );

    

    params.image = (uchar4*)state.d_image;

    hipDeviceptr_t d_param;
    hipMalloc( reinterpret_cast<void**>( &d_param ), sizeof( Params ) );
    hipMemcpy( reinterpret_cast<void*>( d_param ),
        &params, sizeof( params ),
        hipMemcpyHostToDevice );
    
    hipStream_t stream = 0;
    CUDA_CHECK( hipStreamCreate( &stream ) );
    std::chrono::high_resolution_clock::time_point launch_start = std::chrono::high_resolution_clock::now();

    OPTIX_CHECK(optixLaunch( state.pipeline, 
      stream,   // Default CUDA stream
      d_param,
      sizeof( Params ), 
      &state.sbt,
      image_width,
      image_height,
      1 ));
    
    hipFree( (void*)d_param );
    hipDeviceSynchronize();

    torch::Tensor image = torch::from_blob((void*)state.d_image, {image_height, image_width}, torch::TensorOptions().dtype(torch::kInt32).device(torch::kCUDA));
    
    // Stop timer
    std::chrono::high_resolution_clock::time_point launch_end = std::chrono::high_resolution_clock::now();

    // Compute the difference between the two times in milliseconds
    auto launch_time_taken = std::chrono::duration_cast<std::chrono::milliseconds>(launch_end - launch_start).count();
    return image;
}

PYBIND11_MODULE(DiffGaussianRenderer, m) {
    py::class_<OptixState>(m, "OptixState")
        .def(py::init<>());

    m.def("render_gaussians", &render_gaussians, "Render gaussians");
}
